
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>
#include <hipblas.h>

// Define some error checking macros.
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
   if (stat != hipSuccess) {
      fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
   }
}

#define cublasErrCheck(stat) { cublasErrCheck_((stat), __FILE__, __LINE__); }
void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
   if (stat != HIPBLAS_STATUS_SUCCESS) {
      fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
   }
}

#define curandErrCheck(stat) { curandErrCheck_((stat), __FILE__, __LINE__); }
void curandErrCheck_(hiprandStatus_t stat, const char *file, int line) {
   if (stat != HIPRAND_STATUS_SUCCESS) {
      fprintf(stderr, "cuRand Error: %d %s %d\n", stat, file, line);
   }
}

#include <mma.h>
using namespace nvcuda;

// Must be multiples of 16 for wmma code to work
#define MATRIX_M 1024
#define MATRIX_N 1024
#define MATRIX_K 1024

void random_init(int8_t* a, int m, int n) {
    // random initialize matrix A
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            a[i * n + j] = (int8_t) rand() % 127;
        }
    }
}



int main(int argc, char* argv[]) {
   int8_t *a_int8_host, *a_int8_device;
   int8_t *b_int8_host, *b_int8_device;
   int32_t *c_int32_host, *c_int32_device;
   printf("sizeof(int8_t): %lu, sizeof(ing32_t): %lu", sizeof(int8_t), sizeof(int32_t));

   a_int8_host  = (int8_t*) malloc(sizeof(int8_t) * MATRIX_M * MATRIX_K);
   b_int8_host  = (int8_t*) malloc(sizeof(int8_t) * MATRIX_K * MATRIX_N);
   c_int32_host  = (int32_t*) malloc(sizeof(int32_t) * MATRIX_M * MATRIX_N);
   random_init(a_int8_host, MATRIX_M, MATRIX_K);
   random_init(b_int8_host, MATRIX_K, MATRIX_N);

   hipblasHandle_t cublasHandle;
   hipEvent_t startcublas;
   hipEvent_t stopcublas;
   
   cudaErrCheck(hipEventCreate(&startcublas));
   cudaErrCheck(hipEventCreate(&stopcublas));
   cublasErrCheck(hipblasCreate(&cublasHandle));
   // Enable tensor cores
   // cublasErrCheck(cublasSetMathMode(cublasHandle, CUBLAS_DEFAULT_MATH));
   // Disable tensor cores
   cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_PEDANTIC_MATH));
   
   cudaErrCheck(hipMalloc((void**)&a_int8_device, MATRIX_M * MATRIX_K * sizeof(int8_t)));
   cudaErrCheck(hipMalloc((void**)&b_int8_device, MATRIX_K * MATRIX_N * sizeof(int8_t)));
   cudaErrCheck(hipMalloc((void**)&c_int32_device, MATRIX_M * MATRIX_N * sizeof(int32_t)));

   cudaErrCheck(hipMemcpy(a_int8_device, a_int8_host,
      MATRIX_M * MATRIX_K * sizeof(int8_t), hipMemcpyHostToDevice));
   cudaErrCheck(hipMemcpy(b_int8_device, b_int8_host,
      MATRIX_K * MATRIX_N * sizeof(int8_t), hipMemcpyHostToDevice));

   int8_t alpha = 1;
   int8_t beta = 0;
   printf("\nM = %d, N = %d, K = %d. alpha = %d, beta = %d\n\n",
      MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta);
   
   // Now using cuBLAS
   printf("Running with cuBLAS...\n");
   float average_time = 0.0;
   int n_iter = 10000;
   for(int i=0; i<n_iter; i++) {
      float elapse_time = 0.0;
      cudaErrCheck(hipMemset(c_int32_device, 0, MATRIX_M * MATRIX_N * sizeof(int32_t)));
      cudaErrCheck(hipEventRecord(startcublas));
      cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, 
                  MATRIX_M, MATRIX_N, MATRIX_K, 
                  &alpha,
                  a_int8_device, HIP_R_8I, MATRIX_M,
                  b_int8_device, HIP_R_8I, MATRIX_K,
                  &beta, 
                  c_int32_device, HIP_R_32I, MATRIX_M,
                  // CUBLAS_COMPUTE_32I, CUBLAS_GEMM_DEFAULT));  // Enable Tensor Cores
                  HIPBLAS_COMPUTE_32I_PEDANTIC, HIPBLAS_GEMM_DEFAULT));  // Disable Tensor Cores
      cudaErrCheck(hipEventRecord(stopcublas));
      cudaErrCheck(hipEventSynchronize(stopcublas));
      cudaErrCheck(hipEventElapsedTime(&elapse_time, startcublas, stopcublas));
      average_time += elapse_time;
   }
   average_time = average_time / n_iter;
   // Error checking
   // cudaErrCheck(
   //    cudaMemcpy(
   //       c_int32_host, c_int32_device,
   //       MATRIX_M * MATRIX_N * sizeof(int32_t),
   //       cudaMemcpyDeviceToHost
   //    )
   // );
   printf("cublas took %f ms\n", average_time);

   cudaErrCheck(hipEventDestroy(startcublas));             
   cudaErrCheck(hipEventDestroy(stopcublas));
   
   cudaErrCheck(hipFree(a_int8_device));
   cudaErrCheck(hipFree(b_int8_device));
   cudaErrCheck(hipFree(c_int32_device));
   
   free(a_int8_host);
   free(b_int8_host);
   free(c_int32_host);
   cudaErrCheck(hipDeviceReset());
   return 0;
}


